#include <assert.h>
#include <iostream>
#include "manager.hh"
#include "kernel.cu"
using namespace std;

GPUMatAdd::GPUMatAdd(int *A, int *B, int *C, int _x, int _y) {
    // assign host address
    h_A = A;
    h_B = B;
    h_C = C;
    x = _x;
    y = _y;
    int noElements = x * y;
    int bytes = noElements * sizeof(int);
    // alloc device memory
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);
    // transfer from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    
    hipError_t err = hipGetLastError();
    assert(err == 0);
}

void GPUMatAdd::add() {
    int noElements = x * y;
    int bytes = noElements * sizeof(int);

    dim3 block(32, 32);
    dim3 grid((x + block.x - 1) / block.x,
                (y + block.y - 1) / block.y);
    kernel_matrix_sum<<<grid, block>>>(d_A, d_B, d_C, x, y);
    
    hipDeviceSynchronize();
    // copy result memory back
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    assert(err == 0);
}

GPUMatAdd::~GPUMatAdd() {
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}